#include "hip/hip_runtime.h"
#include "Tracer.cuh"
#include "TGAImage.h"
#include "Vec.hpp"
#include "Mesh.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
    } \
} while(0)

// Helper function to calculate a ray-triangle intersection using the Möller–Trumbore algorithm.
__device__ bool intersect_triangle(const vec3& ray_origin, const vec3& ray_dir, const vec3& v0, const vec3& v1, const vec3& v2,float& t, float& u, float& v) {
   const float EPSILON = 1e-5f;
   vec3 edge1 = v1 - v0;
   vec3 edge2 = v2 - v0;
   vec3 h = cross(ray_dir, edge2);
   float a = dot(edge1, h);
   if (a > -EPSILON && a < EPSILON)
      return false;
   float f = 1.0f / a;
   vec3 s = ray_origin - v0;
   u = f * dot(s, h);
   if (u < 0.0f || u > 1.0f)
      return false;
   vec3 q = cross(s, edge1);
   v = f * dot(ray_dir, q);
   if (v < 0.0f || u + v > 1.0f)
      return false;
   t = f * dot(edge2, q);
   if (t > EPSILON) {
      return true;
   }
   return false;
}
// for debugging if needed
__device__ void printf_mat4(const char* name,const mat4& m) {
   printf("%s:\n", name);
   for (int i = 0; i < 4; ++i) {
      printf("[%f, %f, %f, %f]\n", m[i][0], m[i][1], m[i][2], m[i][3]);
   }
}

// The CUDA kernel. This function runs on the GPU.
// Each thread handles one pixel.
__global__ void renderKernel(unsigned char* d_image_data, int width, int height, const DeviceMesh* d_meshes,int num_meshes,const Light* d_lights,int num_lights,const mat4* d_modelview,const mat4* d_projection) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x >= width || y >= height) {
      return;
   }
   // if(x!=400 || y!=300) return;

   mat4 modelview=*d_modelview;
   mat4 projection=*d_projection;
   mat4 inv_modelview = modelview.invert();
   mat4 inv_projection = projection.invert();
   
   // Convert pixel coordinates to NDC space [-1, 1]
   vec4 ray_dir_ndc = {
      (2.0f * (x + 0.5f) / width - 1.0f),
      (2.0f * (y + 0.5f) / height - 1.0f),
      -1.0f, 1.0f
   };

   // Transform ray direction from screen space to world space
   vec4 eye_h=inv_projection*ray_dir_ndc;
   vec3 ray_dir_world = normalized(convert_to_size<3>(inv_modelview* vec4{eye_h.x,eye_h.y,-1.0f,0.0f}));
   vec3 ray_origin_world = convert_to_size<3>(inv_modelview * vec4{0, 0, 0, 1}); //tranpose becuase mat4 is row-major
  
   Ray ray_world = {ray_origin_world, ray_dir_world};
   int closest_mesh=-1,closest_face=-1;
   float t,u,v;
   float closest_t = 1e10f; // Initialize with a large value
   
   // Iterate through all meshes in the scene
   for (int i = 0; i < num_meshes; ++i) {
      const DeviceMesh& mesh = d_meshes[i];
      // Transform the ray into the mesh's local space
      mat4 inv_modelMatrix = mesh.modelMatrix.invert();
      vec3 ray_origin_local = convert_to_size<3>(inv_modelMatrix * convert_to_size<4>(ray_world.origin, 1.0));
      vec3 ray_dir_local = normalized(convert_to_size<3>(inv_modelMatrix * convert_to_size<4>(ray_world.direction, 0.0)));
      // Iterate through all faces of the mesh
      for (int j = 0; j < mesh.num_faces; ++j) {
         vec3i face = mesh.d_faces_verts[j];
         vec3 v0 = mesh.d_verts[face[0]];
         vec3 v1 = mesh.d_verts[face[1]];
         vec3 v2 = mesh.d_verts[face[2]];
         float temp_u,temp_v;
         if (intersect_triangle(ray_origin_local, ray_dir_local, v0, v1, v2, t, temp_u,temp_v)) {
            if (t < closest_t && t > 0.0f) {
               closest_t = t;
               closest_mesh=i;
               closest_face=j;
               u=temp_u;
               v=temp_v;
         }}}}
   if(closest_face==-1){
      d_image_data[(y * width + x) * 3 + 0] = 0;
      d_image_data[(y * width + x) * 3 + 1] = 0;
      d_image_data[(y * width + x) * 3 + 2] = 0;
      return;
   }

   const DeviceMesh& mesh=d_meshes[closest_mesh];
   mat4 inv_modelMatrix_face=mesh.modelMatrix.invert();
   // Sample diffuse texture
   vec3 tex_color;
   if (mesh.d_diffuse_map_data.d_data != nullptr) {
      vec3i face_uv = mesh.d_faces_uvs[closest_face];
      vec2 uv0 = mesh.d_uvs[face_uv[0]];
      vec2 uv1 = mesh.d_uvs[face_uv[1]];
      vec2 uv2 = mesh.d_uvs[face_uv[2]];
      
      vec2 uv = {
         (1.0f-u-v) * uv0.x + u * uv1.x + v * uv2.x,
         (1.0f-u-v) * uv0.y + u * uv1.y + v * uv2.y
      };
      
      int tex_x = (int)(uv.x * (mesh.d_diffuse_map_data.texture_width -1) +0.5f);
      int tex_y = (int)(uv.y * (mesh.d_diffuse_map_data.texture_height -1) +0.5f);

      
      // Clamp coordinates
      tex_x = max(0, min(tex_x, mesh.d_diffuse_map_data.texture_width - 1));
      tex_y = max(0, min(tex_y, mesh.d_diffuse_map_data.texture_height - 1));
      
      int tex_idx = (tex_y * mesh.d_diffuse_map_data.texture_width + tex_x) * 3;
      tex_color.x = mesh.d_diffuse_map_data.d_data[tex_idx + 0];
      tex_color.y = mesh.d_diffuse_map_data.d_data[tex_idx + 1];
      tex_color.z = mesh.d_diffuse_map_data.d_data[tex_idx + 2];
   } else {
      tex_color = vec3{0,0,0};
   }
   
   
   vec3 point_world=ray_origin_world+closest_t*ray_dir_world;
   vec3 final_color_vec = {0.0f, 0.0f, 0.0f};
   
   // Get interpolated normal
   vec3i face_n = mesh.d_faces_norms[closest_face];
   vec3 n0 = mesh.d_norms[face_n[0]];
   vec3 n1 = mesh.d_norms[face_n[1]];
   vec3 n2 = mesh.d_norms[face_n[2]]; 
   vec3 interpolated_normal = normalized(n0 * u + n1 * v + n2 * (1.0f - u - v));
   
      // caluclate light
      for(int li=0;li<num_lights;li++){
         vec3 light_pos_world=d_lights[li].pos;
         float intensity=d_lights[li].intensity;
         
         // shadow rays:
         bool in_shadow=false;
         for (int i = 0; !in_shadow && i < num_meshes; ++i) {
         const DeviceMesh& mesh = d_meshes[i];
         // Transform the ray into the mesh's local space
         mat4 inv_modelMatrix = mesh.modelMatrix.invert();
         vec3 shadow_ray_origin_local = convert_to_size<3>(inv_modelMatrix * convert_to_size<4>(point_world, 1.0));
         vec3 light_pos_local = convert_to_size<3>(inv_modelMatrix * convert_to_size<4>(light_pos_world, 1.0));
         vec3 shadow_ray_dir_local = normalized(light_pos_local - shadow_ray_origin_local);
         float t_shadow;
         float light_dist=norm(light_pos_world-point_world);
         // Iterate through all faces of the mesh
         for (int j = 0; !in_shadow && j < mesh.num_faces; ++j) {
            vec3i face = mesh.d_faces_verts[j];
            vec3 v0 = mesh.d_verts[face[0]];
            vec3 v1 = mesh.d_verts[face[1]];
            vec3 v2 = mesh.d_verts[face[2]];
            float temp_u,temp_v;
            if (intersect_triangle(shadow_ray_origin_local, shadow_ray_dir_local, v0, v1, v2, t_shadow, temp_u,temp_v)) {
               // if(t_shadow > 1e-4f && t_shadow < light_dist)
               in_shadow=true;
            }}}

         if(!in_shadow){
            vec3 light_dir_local = normalized(convert_to_size<3>(inv_modelMatrix_face * convert_to_size<4>(light_pos_world - point_world, 0.0)));
            float diffuse = fmaxf(dot(interpolated_normal, light_dir_local), 0.0f);
            final_color_vec = final_color_vec + tex_color * diffuse * intensity;
         }
      }
      
   // final_color_vec = tex_color; // color of texture
   // repair bgr to rgb
   float temp=final_color_vec.x;
   final_color_vec.x=final_color_vec.z;
   final_color_vec.z=temp;

   vec3 final_color=final_color_vec;

   
   
   // Convert final color to TGAColor format and write to image buffer
   d_image_data[(y * width + x) * 3 + 0] = static_cast<unsigned char>(fminf(final_color.x, 255.0f));
   d_image_data[(y * width + x) * 3 + 1] = static_cast<unsigned char>(fminf(final_color.y, 255.0f));
   d_image_data[(y * width + x) * 3 + 2] = static_cast<unsigned char>(fminf(final_color.z, 255.0f));
}

// Global function to be called from the host (CPU) code
// This is the entry point for CUDA rendering.
void render_cuda(TGAImage& image, const Scene& scene) {
   // Prepare data on the host (CPU) ---
   int width = image.width();
   int height = image.height();
   std::vector<unsigned char> h_image_data(width * height * 3); // Host-side image data
  
   // Create a vector of DeviceMesh structs on the host
   std::vector<DeviceMesh> h_device_meshes(scene.Meshes.size());
   // Pointers for device memory
   DeviceMesh* d_meshes;
   Light* d_lights;
   unsigned char* d_image_data;
   mat4 *d_modelview;
   mat4 *d_projection;

   // Copy data to the device (GPU) ---
   // Allocate device memory for the image
   CHECK_CUDA(hipMalloc(&d_image_data, h_image_data.size()));
   // Allocate device memory for the meshes
   CHECK_CUDA(hipMalloc(&d_meshes, scene.Meshes.size() * sizeof(DeviceMesh)));
   // lights
   CHECK_CUDA(hipMalloc(&d_lights, scene.lights.size() * sizeof(Light)));
   CHECK_CUDA(hipMemcpy(d_lights,scene.lights.data(),scene.lights.size()*sizeof(Light),hipMemcpyHostToDevice));
   // copy matricies
   CHECK_CUDA(hipMalloc(&d_modelview,sizeof(mat4))); 
   CHECK_CUDA(hipMalloc(&d_projection,sizeof(mat4))); 
   CHECK_CUDA(hipMemcpy(d_modelview,&scene.modelview,sizeof(mat4),hipMemcpyHostToDevice ));
   CHECK_CUDA(hipMemcpy(d_projection,&scene.projection,sizeof(mat4),hipMemcpyHostToDevice ));
   int mesh_idx = 0;
   for (const auto& pair : scene.Meshes) {
      const auto& mesh = pair.second;
      DeviceMesh& dmesh=h_device_meshes[mesh_idx];
      // Mesh matrix
      // CHECK_CUDA(hipMalloc(&h_))
      // Copy vertex data
      CHECK_CUDA(hipMalloc(&dmesh.d_verts, mesh->nverts() * sizeof(vec3)));
      CHECK_CUDA(hipMemcpy(dmesh.d_verts, mesh->verts.data(), mesh->nverts() * sizeof(vec3), hipMemcpyHostToDevice));
      // Copy face data
      CHECK_CUDA(hipMalloc(&dmesh.d_faces_verts, mesh->nfaces() * sizeof(vec3i)));
      CHECK_CUDA(hipMemcpy(dmesh.d_faces_verts, mesh->faces_verts.data(), mesh->nfaces() * sizeof(vec3i), hipMemcpyHostToDevice));
      // Copy UV data
      CHECK_CUDA(hipMalloc(&dmesh.d_uvs, mesh->uvs.size() * sizeof(vec2)));
      CHECK_CUDA(hipMemcpy(dmesh.d_uvs, mesh->uvs.data(), mesh->uvs.size() * sizeof(vec2), hipMemcpyHostToDevice));
      // Copy UV face
      CHECK_CUDA(hipMalloc(&dmesh.d_faces_uvs, mesh->faces_uvs.size() * sizeof(vec3i)));
      CHECK_CUDA(hipMemcpy(dmesh.d_faces_uvs, mesh->faces_uvs.data(), mesh->faces_uvs.size() * sizeof(vec3i), hipMemcpyHostToDevice));
      // Copy normal data
      CHECK_CUDA(hipMalloc(&dmesh.d_norms, mesh->norms.size() * sizeof(vec3)));
      CHECK_CUDA(hipMemcpy(dmesh.d_norms, mesh->norms.data(), mesh->norms.size() * sizeof(vec3), hipMemcpyHostToDevice));
      // Copy normal face
      CHECK_CUDA(hipMalloc(&dmesh.d_faces_norms, mesh->faces_norms.size() * sizeof(vec3i)));
      CHECK_CUDA(hipMemcpy(dmesh.d_faces_norms, mesh->faces_norms.data(), mesh->faces_norms.size() * sizeof(vec3i), hipMemcpyHostToDevice));

      // Copy texture data to device
      for(auto mattemp: mesh->materials){
         auto material=mattemp.second;
         TGAImage default_diffuse = material->diffuse_map;
         CHECK_CUDA(hipMalloc(&h_device_meshes[mesh_idx].d_diffuse_map_data.d_data, default_diffuse.width() * default_diffuse.height() * 3));
         CHECK_CUDA(hipMemcpy(h_device_meshes[mesh_idx].d_diffuse_map_data.d_data, default_diffuse.get(), default_diffuse.width() * default_diffuse.height() * 3, hipMemcpyHostToDevice));
         h_device_meshes[mesh_idx].d_diffuse_map_data.texture_width = default_diffuse.width();
         h_device_meshes[mesh_idx].d_diffuse_map_data.texture_height = default_diffuse.height();
         break;
      }
   

      // Set other properties
      h_device_meshes[mesh_idx].num_verts = mesh->nverts();
      h_device_meshes[mesh_idx].num_faces = mesh->nfaces();
      h_device_meshes[mesh_idx].num_uvs = mesh->uvs.size();
      h_device_meshes[mesh_idx].num_norms = mesh->norms.size();
      h_device_meshes[mesh_idx].modelMatrix = mesh->getModelMatrix();

      mesh_idx++;
    }

   // Copy the array of DeviceMesh structs to the device
   CHECK_CUDA(hipMemcpy(d_meshes, h_device_meshes.data(), scene.Meshes.size() * sizeof(DeviceMesh), hipMemcpyHostToDevice));

   // --- Step 3: Launch the kernel ---
   dim3 blockSize(16, 16);
   dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
   renderKernel<<<gridSize, blockSize>>>(d_image_data, width, height, d_meshes, scene.Meshes.size(),d_lights,scene.lights.size(), d_modelview, d_projection);
   CHECK_CUDA(hipGetLastError()); // Check for kernel launch errors
   CHECK_CUDA(hipDeviceSynchronize()); // Wait for the GPU to finish

   
   // --- Step 4: Copy the result back to the host ---
   CHECK_CUDA(hipMemcpy(h_image_data.data(), d_image_data, h_image_data.size(), hipMemcpyDeviceToHost));
    
   // --- Step 5: Clean up and save image ---
   // Free device memory
   CHECK_CUDA(hipFree(d_image_data));
   for (long unsigned int i = 0; i < h_device_meshes.size(); i++) {
      CHECK_CUDA(hipFree(h_device_meshes[i].d_verts));
      CHECK_CUDA(hipFree(h_device_meshes[i].d_faces_verts));
      CHECK_CUDA(hipFree(h_device_meshes[i].d_uvs));
      CHECK_CUDA(hipFree(h_device_meshes[i].d_faces_uvs));
      CHECK_CUDA(hipFree(h_device_meshes[i].d_norms));
      CHECK_CUDA(hipFree(h_device_meshes[i].d_faces_norms));
      if (h_device_meshes[i].d_diffuse_map_data.d_data != nullptr) {
         CHECK_CUDA(hipFree(h_device_meshes[i].d_diffuse_map_data.d_data));
      }
   }
   CHECK_CUDA(hipFree(d_lights));
   CHECK_CUDA(hipFree(d_meshes));
   CHECK_CUDA(hipFree(d_projection));
   CHECK_CUDA(hipFree(d_modelview));

   // Convert the raw data to a TGA image
   for (int y = 0; y < height; ++y) {
      for (int x = 0; x < width; ++x) {
         int idx = (y * width + x) * 3;
         TGAColor color(h_image_data[idx], h_image_data[idx+1], h_image_data[idx+2]);
         image.set(x, y, color);
      }
   }
}

